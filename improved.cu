#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define PI 3.142857

const int gridSize = 4;
const int mx = 256;
const int my = 256;

typedef struct {
    double x, y;
} vector2;

__device__ vector2 findNearestCorner(int x, int y) {

    double xOffset, yOffset;

    xOffset = x % gridSize;
    yOffset = y % gridSize;

    double bx, by;
    
    bx = xOffset <= gridSize/2;
    by = yOffset <= gridSize/2;

    vector2 v;

    if (bx) {
        if (by) {
            v.x = x - xOffset + gridSize;
            v.y = y - yOffset + gridSize;
        }
        else {
            v.x = x - xOffset + gridSize;
            v.y = y - yOffset;
        }
    }
    else {
        if (by) {
            v.x = x - xOffset;
            v.y = y - yOffset + gridSize;
        }
        else {
            v.x = x - xOffset;
            v.y = y - yOffset;
        }
    }
    return v;
}

__global__ void perlinNoise(int mx, int my, int mapx, int mapy, double *deviceVectors, double *out) {
    
    int threadId = blockIdx.x*blockDim.x+threadIdx.x;
    if (threadId > (mx*my)){
    }
    else{
    	int x = (threadId % my);
    	
    	int y = int(threadId / my); 
    	
    	vector2 corner, offSetVector, cornerVector;
	
    	corner = findNearestCorner(x, y);
	
    	offSetVector.x = corner.x - ((double)x);
    	offSetVector.y = corner.y - ((double)y);

    	double cornerVectorAngle, value;
    
    	int vectorIndex = (int)(corner.y / gridSize)*((corner.x / gridSize) + 2);
    
    	cornerVectorAngle = (deviceVectors)[vectorIndex];
    	//printf("wtf %i\n",threadIdx.x);
        
        
    	cornerVector.y = sin(cornerVectorAngle);
    	cornerVector.x = cos(cornerVectorAngle);
        
        
    	value = (offSetVector.x*cornerVector.x + offSetVector.y*cornerVector.y);
    	value = ((value + 1)/2);
    
    	//printf("%f %f %f %f %f %f %f %d %d %f %d\n", value, offSetVector.x, offSetVector.y, cornerVector.x, cornerVector.y, corner.x, corner.y, x, y, cornerVectorAngle, vectorIndex);
        
    	(out)[threadId] = value;
    	//printf("%f \n", value);
    }
}

int main() {
    
    srand(141414);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    int mapx, mapy, nDot;
    
    mapx = ceil(((double)mx)/gridSize);
    mapy = ceil(((double)my)/gridSize);

    nDot = (mapx + 1)*(mapy + 1);

    printf("mx=%i\nmy=%i\nmapx=%i\nmapy=%i\nnDot=%i\n",mx,my,mapx,mapy,nDot);

    double* vectors = (double*) malloc(nDot * sizeof(double));
    double* output = (double*) malloc((mx * my) * sizeof(double));

    for (int i = 0; i < nDot; i++) {
        vectors[i] = rand() * 2 * PI / RAND_MAX;
        //printf("%i\t%f\n", i, vectors[i]);
    }
    
    double *deviceVectors = NULL;
    hipMalloc((void **)&deviceVectors, (nDot * sizeof(double)));
    
    double *deviceOutput = NULL;
    hipMalloc((void **)&deviceOutput, ((mx * my) * sizeof(double)));
    
    //hipMemcpy(deviceVectors, vectors, (nDot * sizeof(double)), hipMemcpyHostToDevice);
    
    int threadsPerBlock = 256;
    int blocksPerGrid = ceil((mx * my) / threadsPerBlock); 
    
    // start
    
    hipEventRecord(start);
    
    const int numStreams = 8;
    hipStream_t streams[numStreams];
    
    for (int i = 0; i < numStreams; i++){
    	hipStreamCreate(&(streams[i]));
    }
    
    int chunkSize = ceil((mx*my) / numStreams);
    
    for (int stream = 0; stream < numStreams; stream++){
    const int lower = chunkSize * stream;
    const int upper = min(lower + chunkSize , mx*my);
    const int width = upper - lower;
    
    hipMemcpyAsync(deviceVectors + lower, vectors + lower, sizeof(double)*width, hipMemcpyHostToDevice, streams[stream]);
    
    perlinNoise<<<blocksPerGrid, threadsPerBlock, 0, streams[stream]>>>(mx, my, mapx, mapy, deviceVectors + lower, deviceOutput);
    
    hipMemcpyAsync(output + lower, deviceOutput + lower, sizeof(double)*width, hipMemcpyDeviceToHost, streams[stream]);
    }
    // end

    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Elapsed time: %f milliseconds\n", milliseconds);
    
    //perlinNoise<<<blocksPerGrid, threadsPerBlock>>>(mx, my, mapx, mapy, deviceVectors, deviceOutput);
    
    
    //hipMemcpy(output, deviceOutput, ((mx * my) * sizeof(double)), hipMemcpyDeviceToHost);
    
    
    FILE *fptr;

    fptr = fopen("output.txt","w");

    for (int i = 0; i < my; i++){
    	for (int j = 0; j < mx; j++){
    		//printf("%f ", output[j+(i*j)]);
    		fprintf(fptr, "%f ", output[j+(i*j)]);
    	}
    	//printf("\n");
    	fprintf(fptr, "%s", "\n");
    }
    
    fclose(fptr);
    

    return 0;
}
